// System includes
#include <stdio.h>
#include <assert.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#include <chrono>
using namespace std::chrono;


__global__ void gpu_add_fun(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}


int local_add(int N, int *a, int *b, int *c) {

  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  for (int i=0; i<N; i++) c[i] = a[i] + b[i];
  high_resolution_clock::time_point t2 = high_resolution_clock::now();

  auto duration = duration_cast<microseconds>( t2 - t1 ).count();
  printf("\nLocal:            Elapsed time %f: msec.   ", duration/1000.0f);

  long s = 0;
  for (int i=0; i<N; i++) s += (c[i]);

  return s;
}


int random_ints(int *x, int N) {

  srand (time(NULL));
  for (int i = 0; i<N; i++) x[i] = (rand() % 3 - 1);

  return 0;
}


int gpu_add(int N, int m, int *a, int *b, int *c, int *c_gpu) {

  // for measuring execution time
  hipError_t error;
  hipEvent_t start;
  error = hipEventCreate(&start);
  hipEvent_t stop;
  error = hipEventCreate(&stop);

  int *d_a, *d_b, *d_c;					// gpu variables
  int size = N * sizeof(int);				// allocation space size

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  error = hipEventRecord(start, 0);

  // Launch add() kernel on GPU with N blocks 
  gpu_add_fun<<<N/m,m>>>(d_a, d_b, d_c);
  
  error = hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  
  // calculate execution time
  float msecTotal = 0.0f;
  error = hipEventElapsedTime(&msecTotal, start, stop);
  printf("Elapsed time %f: msec.   ", msecTotal);

  // Copy result back to host
  hipMemcpy(c_gpu, d_c, size, hipMemcpyDeviceToHost);

  // Variable cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  long s = 0;
  for (int i=0; i<N; i++) s += (c_gpu[i]);

  return s;
}



int main(void)
{
  // timer
  int N = 1<<15; //2^15
  int size = N * sizeof(int);

  int *a, *b, *c, *c_gpu;				// local variables
  long sum = 0;
  
  // Alloc space for host copies of a, b, c and setup input values 
  a = new int[size]; random_ints(a, N);
  b = new int[size]; random_ints(b, N);
  c = new int[size];
  c_gpu = new int[size];


  sum = local_add(N, a, b, c);
  printf("Sum diff: %ld\n", sum);

  
  for (int i=1; i<=(1<<15); i*=2) {
    printf("GPU i = %5d.    ", i);
    sum = gpu_add(N, i, a, b, c, c_gpu); 
    printf("Sum diff: %ld\n", sum);
  }

  // local variable cleanup
  delete [] a; delete [] b; delete [] c;

  return 0;
}











